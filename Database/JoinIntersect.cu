// Author : Pinaggle
// Last Modified: 20018/11/2 14:38

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <iomanip>
#include <time.h>
using namespace std;

/*
 * this function is used to do multi-join on GPU, the lists to be joined is recorded
 * one by one in arr[],and the start position of each list is recorded in head[].
 * table saves which lists are to be intersected. It's size is join_length*item_size.
 * Each column of table represent a multi-way join, so we hacve item_size results at
 * last.
 */
void disp(const char* name,unsigned *arr,unsigned length,unsigned line)
{
 	cout << name << endl;
	for(unsigned i = 0;i < length;i++)
	{
		cout <<arr[i] << ' ';
		if(i % line == line-1)
			cout << endl;
	}
	cout << endl;
}
void start_clock(struct timespec &start)
{
	clock_gettime(CLOCK_MONOTONIC,&start);
}

void gettime_and_reset_clock(struct timespec &start, const char* info)
{
	struct timespec cnt;
	clock_gettime(CLOCK_MONOTONIC,&cnt);
	double period = 1000*(cnt.tv_sec - start.tv_sec)
			+ 0.000001*(cnt.tv_nsec - start.tv_nsec);
	cout << info << " used " << setiosflags(ios::fixed) 
		<< setprecision(2) << period <<"ms"<<endl;
	start = cnt;
}
void gettime(struct timespec &start, const char* info)
{
	
	struct timespec cnt;
	clock_gettime(CLOCK_MONOTONIC,&cnt);
	double period = 1000*(cnt.tv_sec - start.tv_sec)
			+ 0.000001*(cnt.tv_nsec - start.tv_nsec);
	cout << info << " used " << setiosflags(ios::fixed)
		<< setprecision(2) << period <<"ms"<<endl;
} 
void cudaInit()
{
	unsigned *d_init;
	cout << "CudaInitFinished" << endl;
	hipMalloc(&d_init,sizeof(unsigned));
	hipFree(d_init);
}
__global__ void binarySearch(unsigned *d_head,unsigned *d_arr,unsigned join_width,
		unsigned table_length,unsigned *d_can_list,unsigned can_list_size)
{
	unsigned idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned gridSize = gridDim.x*blockDim.x;

	while(idx < d_head[table_length])
	{
		unsigned key = d_arr[idx];
		unsigned l = 0,r = table_length,m,i;
		while(l+1 < r)
		{
			m = (l+r)/2;
			if(d_head[m] <= idx)
				l = m;
			else
				r = m;
		}
		i = l;
		bool found = true;
		for(unsigned j = 1;j < join_width;j++)
		{
			l = d_head[j*table_length+i];
			r = d_head[j*table_length+i+1];
			while(l+1 < r)
			{
				m = (l+r)/2;
				if(d_arr[m] <= key)
					l = m;
				else 
					r = m;
			}
			if((l == r)||(d_arr[l] != key))
			{
				found = false;
				break;
			}
		}
		if(found && (d_can_list != NULL))
		{
			l = 0;
			r = can_list_size;
			while(l+1 < r)
			{
				m = (l+r)/2;
				if(d_can_list[m] <= key)
					l = m;
				else 
					r = m;
			}
			if((l == r)||(d_can_list[l] != key))
				found = false;
		}
		if(!found)
			d_arr[idx] = -1;
		idx += gridSize;
	}	

}
void calc(unsigned *head,unsigned *arr, unsigned join_width,unsigned table_length,
		unsigned item_num,unsigned *can_list,unsigned can_list_size)
{
#ifdef DEBUG_JOIN
	disp("head", &head[0], head.size(), table_length);
#endif
	// bool use_gpu = false;
	bool use_gpu = true;
	if(use_gpu)
	{
		cout << "enter GPU mode"<<endl;
		unsigned *d_head, *d_arr, *d_can_list = NULL;
		hipMalloc(&d_head, sizeof(unsigned)*(join_width*table_length+1));
		hipMalloc(&d_arr, sizeof(unsigned)*item_num);
		if(can_list != NULL)
			hipMalloc(&d_can_list, sizeof(unsigned)*can_list_size);
		if (d_head == NULL || d_arr == NULL || (can_list != NULL && d_can_list == NULL))
		{
			cout << "GPU malloc failed!" << endl;
			if (d_head) hipFree(d_head);
			if (d_arr) hipFree(d_arr);
			if (d_can_list) hipFree(d_can_list);
			goto CPUMODE;
		}
		hipMemcpy(d_head, head, sizeof(unsigned)*(join_width*table_length+1), hipMemcpyHostToDevice);
		hipMemcpy(d_arr, arr, sizeof(unsigned)*item_num, hipMemcpyHostToDevice);
		if(can_list != NULL)
			hipMemcpy(d_can_list, can_list, sizeof(unsigned)*can_list_size, hipMemcpyHostToDevice);
		int threadsPerBlock = 256;
		int blocksPerGrid = (head[table_length] + threadsPerBlock - 1) / threadsPerBlock;
		if(blocksPerGrid > 1024)
			blocksPerGrid = 1024;
		hipEvent_t start,stop;
		float period;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start,0);
		binarySearch <<<blocksPerGrid, threadsPerBlock >>> (d_head,d_arr,
				join_width,table_length,d_can_list,can_list_size);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&period,start,stop);
		printf("****Kernel Execution time: %.2f ms\n",period);
		hipMemcpy(arr, d_arr, sizeof(unsigned)*head[table_length], hipMemcpyDeviceToHost);
		hipFree(d_head);
		hipFree(d_arr);
		if(can_list != NULL)
			hipFree(d_can_list);
	}
	else
	{
CPUMODE:
		cout << "enter CPU mode" << endl;
		struct timespec start;
		for (unsigned i = 0; i < table_length; i++)
			for(unsigned idx = head[i];idx < head[i+1];idx++) 
		{
			unsigned key = arr[idx];
			unsigned l,r,m;
			bool found = true;
			for (unsigned j = 1; j < join_width; j++)
			{
				l = head[j*table_length+i];
				r = head[j*table_length+i+1];
				while (l + 1 < r)
				{
					m = (l + r) / 2;
					if (arr[m] <= key)
						l = m;
					else
						r = m;
				}
				if ((l == r) || (arr[l] != key))
				{
					found = false;
					break;
				}
			}
			if(found && can_list != NULL)
			{
				l = 0;
				r = can_list_size;
				while (l + 1 < r)
				{
					m = (l + r) / 2;
					if (can_list[m] <= key)
						l = m;
					else
						r = m;
				}
				if ((l == r) || (can_list[l] != key))
				{
					found = false;
				}
			}
			if(!found)
				arr[idx] = -1;			
		}
	}
}
